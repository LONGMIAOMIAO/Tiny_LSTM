
#include <hip/hip_runtime.h>
#include <stdio.h>
#define N 64
#define TPB 32

float scale( int i, int n )
{
    return ( (float)i / (n - 1) );
}

__device__ float distance( float x1, float x2 )
{
    return sqrt( (x2 - x1) * ( x2 - x1) );
}

__global__ void distanceKernel( float* d_out, float* d_in, float ref )
{
    const int i = blockIdx.x * blockDim.x + threadIdx.x;
    d_out[i] = distance(d_in[i], ref);
    printf( "%f\n", d_out[i] );
}

int main()
{
    const float ref = 0.5f;

    float* in = 0;
    float* out = 0;

    hipMallocManaged( &in, N* sizeof(float) );
    hipMallocManaged( &out, N* sizeof(float) );

    for(int i = 0; i < N; i++)
    {
        in[i] = scale(i, N);
    }

    distanceKernel<<<N/TPB, TPB>>>( out, in, ref );

    hipFree(in);
    hipFree(out);
    return 0;
}