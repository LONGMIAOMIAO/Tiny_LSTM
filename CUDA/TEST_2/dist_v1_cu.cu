
#include <hip/hip_runtime.h>
#include <stdio.h>
#define N 64
#define TPB 64

//  Creating Data to be Calculated
__device__ float scale( int i, int n )
{
    return ((float)i) / (n - 1);
}
//  Calculating Distance Between X1 And X2
__device__ float distance( float x1, float x2 )
{
    return sqrt( (x1 - x2)* (x1 - x2) );
}
//  Caculating By CUDA WOWOWOWOW!
__global__ void distance_Kernel( float* d_out, float ref, int len )
{
    const int i = blockIdx.x * blockDim.x + threadIdx.x;
    const float x = scale(i, len);
    d_out[i] = distance(x,ref);
    printf( "%f\n",d_out[i] );
    //std::cout << d_out[i] << std::endl;
}

int main()
{
    const float ref = 0.5f;
    float* d_out = 0;

    hipMalloc(&d_out,N*sizeof(float));

    distance_Kernel<<<N/TPB,TPB>>>(d_out,ref,N);

    hipFree(d_out);
    return 0;
}