#include "hip/hip_runtime.h"
#include "kernel.h"
#include <stdio.h>
#include <vector>
#define TPB 32

__device__ float distance( float x1, float x2 )
{
    return sqrt( (x2 - x1) * ( x2 - x1) );
}

__global__ void distanceKernel( float* d_out, float* d_in, float ref )
{
    const int i = blockIdx.x * blockDim.x + threadIdx.x;
    d_out[i] = distance(d_in[i], ref);
    printf( "%f\n", d_out[i] );
}

void distanceArray( float* out, float* in, float ref, int n )
{
    std::vector<int> vec;

    float* d_in = 0;
    float* d_out = 0;

    hipMalloc( &d_in, n*sizeof(float) );
    hipMalloc( &d_out, n*sizeof(float) );

    hipMemcpy( d_in, in, n*sizeof( float ), hipMemcpyHostToDevice );

    distanceKernel<<<n/TPB, TPB>>>( d_out, d_in, ref );

    hipMemcpy( out, d_out, n*sizeof(float), hipMemcpyDeviceToHost );

    hipFree( d_in );
    hipFree( d_out );
}