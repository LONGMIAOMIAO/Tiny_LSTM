#include "hip/hip_runtime.h"
#include <stdio.h>
#include <math.h>

template <typename T>
struct Mat
{
    int width;
    int height;
    T* elements;
};

template <typename T>
__device__ 
T getElement( Mat<T> *A, int row, int col )
{
    return A->elements[ row * A->width + col ];
}

template <typename T>
__device__
void setElement( Mat<T>* A, int row, int col, T value )
{
    A->elements[ row * A->width + col ] = value;
}

template <typename T>
__global__
void matMulKernel(Mat<T> *A, Mat<T> *B, Mat<T> *C)
{
    T cvalue = 0;

	int row = threadIdx.y + blockIdx.y * blockDim.y;
	int col = threadIdx.x + blockIdx.x * blockDim.x;
	for (int i = 0; i < A->width; ++i)
	{
		cvalue += getElement(A, row, i) * getElement(B, i, col);
	}
	setElement(C, row, col, cvalue);
}

void T_1()
{
    Mat<float>* left;
    Mat<float>* w;
    Mat<float>* out;

    hipMallocManaged( &left, sizeof(Mat<float>) );
    hipMallocManaged( &w, sizeof(Mat<float>) );
    hipMallocManaged( &out, sizeof(Mat<float>) );

    hipMallocManaged( &left->elements, 784 * sizeof(float) );
    hipMallocManaged( &w->elements, 784 * 10 * sizeof(float) );
    hipMallocManaged( &out->elements, 10 * sizeof(float) );

    left->width = 784;
    left->height = 1;

    w->width = 10;
    w->height = 784;

    out->width = 10;
    out->height = 1;

    float s = 1;
    for( int i = 0; i < 784; i++ )
    {
        left->elements[i] = s;
    }

    float t = 1;
    for( int i = 0; i < 784*10; i++ )
    {
        w->elements[i] = t;
    }


    dim3 DimGrid(1, 1, 1);
    dim3 DimBlock(10, 1, 1);

    matMulKernel<float><<< DimGrid, DimBlock >>>( left, w, out );

    hipDeviceSynchronize();

    float num = 0;
    for( int i = 0; i < 10; i++ )
    {
        if( out->elements[i] != 78.4 )
        {
            num++;
        }
    }

    printf( "%f\n", out->elements[0] );
    printf( "%f\n", out->elements[1] );
    printf( "%f\n", out->elements[2] );
    printf( "%f\n", out->elements[9] );

    printf( "%f\n", num );
}

void T_2()
{
    Mat<float>* left;
    Mat<float>* w;
    Mat<float>* out;

    hipMallocManaged( &left, sizeof(Mat<float>) );
    hipMallocManaged( &w, sizeof(Mat<float>) );
    hipMallocManaged( &out, sizeof(Mat<float>) );

    hipMallocManaged( &left->elements, 100 * 100 * 1024 * sizeof(float) );
    hipMallocManaged( &w->elements,    100 * 100 * 1024 * sizeof(float) );
    hipMallocManaged( &out->elements,  100 * 100 * 1024 * sizeof(float) );

    left->width =   100 * 32;
    left->height =  100 * 32;

    w->width =  100 * 32;
    w->height = 100 * 32;

    out->width =  100 * 32;
    out->height = 100 * 32;

    float s = 0.1;
    for( int i = 0; i < 100 * 100 * 1024; i++ )
    {
        left->elements[i] = s;
        //s++;
    }

    float t = 0.1;
    for( int i = 0; i < 100 * 100 * 1024; i++ )
    {
        w->elements[i] = t;
        //t++;
    }

    dim3 DimGrid( 100, 100, 1);
    dim3 DimBlock(32, 32, 1);

    matMulKernel<float><<< DimGrid, DimBlock >>>( left, w, out );

    hipDeviceSynchronize();

    float num = 0;
    for( int i = 0; i < 100 * 100 *1024 ; i++ )
    {
        if( abs ( out->elements[i] - 32 ) > 1 )
        {
            num++;
        }
    }

    printf( "%f\n", out->elements[0] );
    printf( "%f\n", out->elements[1] );
    printf( "%f\n", out->elements[2] );
    printf( "%f\n", out->elements[100 * 100 *1024 - 3 ] );
    printf( "%f\n", out->elements[100 * 100 *1024 - 2 ] );
    printf( "%f\n", out->elements[100 * 100 *1024 - 1 ] );

    printf( "%f\n", num );
}

int main()
{
    auto start = clock();
    //T_1();
    //T_2();
    //T_1();
    T_2();
    auto end = clock();
    printf( "%lf", (end - start) / 1000000.0 );
    return 0;
}