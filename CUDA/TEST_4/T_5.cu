
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>

template <typename T>
struct Mat
{
    int width;
    int height;
    T* elements;
};

template <typename T>
__device__ 
T getElement( Mat<T> *A, int row, int col )
{
    return A->elements[ row * A->width + col ];
}

template <typename T>
__device__
void setElement( Mat<T>* A, int row, int col, T value )
{
    A->elements[ row * A->width + col ] = value;
}

template <typename T>
__global__
void matMulKernel(Mat<T> *A, Mat<T> *B, Mat<T> *C)
{
    T cvalue = 0;

	int row = threadIdx.y + blockIdx.y * blockDim.y;
	int col = threadIdx.x + blockIdx.x * blockDim.x;
	for (int i = 0; i < A->width; ++i)
	{
		cvalue += getElement(A, row, i) * getElement(B, i, col);
	}
	setElement(C, row, col, cvalue);
}

void T_1()
{
    Mat<float>* left;
    Mat<float>* w;
    Mat<float>* out;

    hipMallocManaged( &left, sizeof(Mat<float>) );
    hipMallocManaged( &w, sizeof(Mat<float>) );
    hipMallocManaged( &out, sizeof(Mat<float>) );

    hipMallocManaged( &left->elements, 2 * 2 * 30 * 30 * sizeof(float) );
    hipMallocManaged( &w->elements, 2 * 2 * 30 * 30 * sizeof(float) );
    hipMallocManaged( &out->elements, 2 * 2 * 30 * 30 * sizeof(float) );

    left->width = 60;
    left->height = 60;

    w->width = 60;
    w->height = 60;

    out->width = 60;
    out->height = 60;

    float s = 0;
    for( int i = 0; i < 2 * 2 * 30 * 30; i++ )
    {
        left->elements[i] = s;
        w->elements[i] = s;

        s += 1;
    }

    dim3 DimGrid(2, 2, 1);
    dim3 DimBlock(30, 30, 1);

    matMulKernel<float><<< DimGrid, DimBlock >>>( left, w, out );

    hipDeviceSynchronize();

    printf( "%f\n", out->elements[0] );
    printf( "%f\n", out->elements[1] );
    printf( "%f\n", out->elements[2] );
    printf( "%f\n", out->elements[3599] );
    //printf( "%f\n", out->elements[3600] );
}

void T_2()
{
    Mat<float>* left;
    Mat<float>* w;
    Mat<float>* out;

    hipMallocManaged( &left, sizeof(Mat<float>) );
    hipMallocManaged( &w, sizeof(Mat<float>) );
    hipMallocManaged( &out, sizeof(Mat<float>) );

    hipMallocManaged( &left->elements, 2 * 2 * 30 * 30 * sizeof(float) );
    hipMallocManaged( &w->elements, 2 * 2 * 30 * 30 * sizeof(float) );
    hipMallocManaged( &out->elements, 2 * 2 * 30 * 30 * sizeof(float) );

    left->width = 60;
    left->height = 60;

    w->width = 60;
    w->height = 60;

    out->width = 60;
    out->height = 60;

    float s = 1;
    for( int i = 0; i < 2 * 2 * 30 * 30; i++ )
    {
        left->elements[i] = s;
        w->elements[i] = s;

        //s += 1;
    }

    dim3 DimGrid(2, 2, 1);
    dim3 DimBlock(30, 30, 1);

    matMulKernel<float><<< DimGrid, DimBlock >>>( left, w, out );

    matMulKernel<float><<< DimGrid, DimBlock >>>( out, w, left );


    hipDeviceSynchronize();

    float num = 0;
    for( int i = 0; i < 3600; i++ )
    {
        if( left->elements[i] != 3600 )
        {
            num++;
        }
    }

    printf( "%f\n", out->elements[0] );
    printf( "%f\n", out->elements[1] );
    printf( "%f\n", out->elements[2] );
    printf( "%f\n", out->elements[1589] );
    printf( "%f\n", out->elements[3599] );

    printf( "%f\n", left->elements[0] );
    printf( "%f\n", left->elements[1] );
    printf( "%f\n", left->elements[2] );

    printf( "%f\n", left->elements[3599] );
    printf( "%f\n", left->elements[1589] );
    //printf( "%f\n", out->elements[3600] );
    printf( "%f\n", 3333333333333.0 );
    printf( "%f\n", num );
}

void T_3()
{
    Mat<float>* left;
    Mat<float>* w;
    Mat<float>* out;

    hipMallocManaged( &left, sizeof(Mat<float>) );
    hipMallocManaged( &w, sizeof(Mat<float>) );
    hipMallocManaged( &out, sizeof(Mat<float>) );

    hipMallocManaged( &left->elements, 784 * sizeof(float) );
    hipMallocManaged( &w->elements, 784 * 10 * sizeof(float) );
    hipMallocManaged( &out->elements, 10 * sizeof(float) );

    left->width = 784;
    left->height = 1;

    w->width = 10;
    w->height = 784;

    out->width = 10;
    out->height = 1;

    float s = 0.1;
    for( int i = 0; i < 784; i++ )
    {
        left->elements[i] = s;
        //w->elements[i] = s;

        //s += 1;
    }

    float t = 1;
    for( int i = 0; i < 784*10; i++ )
    {
        //left->elements[i] = s;
        w->elements[i] = t;

        //s += 1;
    }


    dim3 DimGrid(1, 1, 1);
    dim3 DimBlock(10, 1, 1);

    matMulKernel<float><<< DimGrid, DimBlock >>>( left, w, out );

    //matMulKernel<float><<< DimGrid, DimBlock >>>( out, w, left );


    hipDeviceSynchronize();

    float num = 0;
    for( int i = 0; i < 10; i++ )
    {
        if( out->elements[i] != 78.4 )
        {
            num++;
        }
    }

    printf( "%f\n", out->elements[0] );
    printf( "%f\n", out->elements[1] );
    printf( "%f\n", out->elements[2] );
    printf( "%f\n", out->elements[3] );
    printf( "%f\n", out->elements[9] );

    printf( "%f\n", 3333333333333.0 );
    printf( "%f\n", num );
}

int main()
{
    //T_1();
    //T_2();
    T_3();
    return 0;
}