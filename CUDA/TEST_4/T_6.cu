
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>

template <typename T>
struct Mat
{
    int width;
    int height;
    T* elements;
};

template <typename T>
__device__ 
T getElement( Mat<T> *A, int row, int col )
{
    return A->elements[ row * A->width + col ];
}

template <typename T>
__device__
void setElement( Mat<T>* A, int row, int col, T value )
{
    A->elements[ row * A->width + col ] = value;
}

template <typename T>
__global__
void matMulKernel(Mat<T> *A, Mat<T> *B, Mat<T> *C)
{
    T cvalue = 0;

	int row = threadIdx.y + blockIdx.y * blockDim.y;
	int col = threadIdx.x + blockIdx.x * blockDim.x;
	for (int i = 0; i < A->width; ++i)
	{
		cvalue += getElement(A, row, i) * getElement(B, i, col);
	}
	setElement(C, row, col, cvalue);
}

void T_1()
{
    Mat<float>* left;
    Mat<float>* w;
    Mat<float>* out;

    hipMallocManaged( &left, sizeof(Mat<float>) );
    hipMallocManaged( &w, sizeof(Mat<float>) );
    hipMallocManaged( &out, sizeof(Mat<float>) );

    hipMallocManaged( &left->elements, 784 * sizeof(float) );
    hipMallocManaged( &w->elements, 784 * 10 * sizeof(float) );
    hipMallocManaged( &out->elements, 10 * sizeof(float) );

    left->width = 784;
    left->height = 1;

    w->width = 10;
    w->height = 784;

    out->width = 10;
    out->height = 1;

    float s = 1;
    for( int i = 0; i < 784; i++ )
    {
        left->elements[i] = s;
    }

    float t = 1;
    for( int i = 0; i < 784*10; i++ )
    {
        w->elements[i] = t;
    }


    dim3 DimGrid(1, 1, 1);
    dim3 DimBlock(10, 1, 1);

    matMulKernel<float><<< DimGrid, DimBlock >>>( left, w, out );

    hipDeviceSynchronize();

    float num = 0;
    for( int i = 0; i < 10; i++ )
    {
        if( out->elements[i] != 78.4 )
        {
            num++;
        }
    }

    printf( "%f\n", out->elements[0] );
    printf( "%f\n", out->elements[1] );
    printf( "%f\n", out->elements[2] );
    printf( "%f\n", out->elements[9] );

    printf( "%f\n", num );
}

void T_2()
{
    Mat<float>* left;
    Mat<float>* w;
    Mat<float>* out;

    hipMallocManaged( &left, sizeof(Mat<float>) );
    hipMallocManaged( &w, sizeof(Mat<float>) );
    hipMallocManaged( &out, sizeof(Mat<float>) );

    hipMallocManaged( &left->elements, 784 * sizeof(float) );
    hipMallocManaged( &w->elements, 10 * sizeof(float) );
    hipMallocManaged( &out->elements, 784 * 10 * sizeof(float) );

    left->width = 1;
    left->height = 784;

    w->width = 10;
    w->height = 1;

    out->width = 10;
    out->height = 784;

    float s = 1;
    for( int i = 0; i < 784; i++ )
    {
        left->elements[i] = s;
        s++;
    }

    float t = 1;
    for( int i = 0; i < 10; i++ )
    {
        w->elements[i] = t;
        t++;
    }

    dim3 DimGrid(1, 16*7, 1);
    dim3 DimBlock(10, 7, 1);

    matMulKernel<float><<< DimGrid, DimBlock >>>( left, w, out );

    hipDeviceSynchronize();

    float num = 0;
    for( int i = 0; i < 784 * 10; i++ )
    {
        if( out->elements[i] == 0 )
        {
            num++;
        }
    }

    printf( "%f\n", out->elements[0] );
    printf( "%f\n", out->elements[1] );
    printf( "%f\n", out->elements[2] );
    printf( "%f\n", out->elements[784*10-555] );
    printf( "%f\n", out->elements[784*10-3] );
    printf( "%f\n", out->elements[784*10-1] );
    //printf( "%f\n", out->elements[784*10] );

    printf( "%f\n", num );
}

int main()
{
    T_1();
    //T_2();
    //T_1();
    T_2();
    return 0;
}