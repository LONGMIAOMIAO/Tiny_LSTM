
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>
// __global__ 
// void cal_1( double* in, double* out )
// {
//     const int i = blockIdx.x * blockDim.x + threadIdx.x;
//     out[i] = sqrt(sqrt(sqrt( in[i] )));
// }

template <typename T>
__global__
void cal( T* left, T* out )
{
    const int i = blockIdx.x * blockDim.x + threadIdx.x;
    out[i] = left[i] * 3;
}

int main()
{
    float* left;
    float* out;

    hipMallocManaged( &left, 2 * 512 * sizeof(float) );
    hipMallocManaged( &out, 2 * 512 * sizeof(float) );

    float s = 0;
    for( int i = 0; i < 2 * 512; i++ )
    {
        left[i] = i / 2.0 / 512.0 ;
        s = s + left[i];
    }
    printf( "%f\n", s * 3 );

    cal<<< 2, 512 >>>( left, out );

    hipDeviceSynchronize();

    float num = 0;
    for( int i = 0; i < 2* 512; i++ )
    {
        num = num + out[i];
    }

    printf( "%f\n", num );

    return 0;
}