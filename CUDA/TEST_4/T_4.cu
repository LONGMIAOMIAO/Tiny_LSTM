
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>

template <typename T>
__global__
void cal( T* left, T* out, T* w, int left_Row, int left_Col, int w_Row, int w_Col )
{
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    T sum = 0;
    for ( int i = 0; i < left_Col; ++i )
    {
        sum += left[ row * left_Col + i ] * w[ i * w_Col + col ];
    } 
    out[ row * w_Col + col ] = sum;
}

int main()
{
    float* left;
    float* w;
    float* out;

    hipMallocManaged( &left, 2 * 2 * 30 * 30 * sizeof(float) );
    hipMallocManaged( &w, 2 * 2 * 30 * 30 * sizeof(float) );
    hipMallocManaged( &out, 2 * 2 * 30 * 30 * sizeof(float) );

    float s = 1;
    for( int i = 0; i < 2 * 2 * 30 * 30; i++ )
    {
        left[i] = s;
        w[i] = s;
        //s += 1;
    }

    dim3 DimGrid(2, 2, 1);
    dim3 DimBlock(30, 30, 1);

    cal<<< DimGrid, DimBlock >>>( left, out, w, 30 * 2, 30 * 2, 30 * 2, 30 * 2 );


    hipDeviceSynchronize();

    float num = 0;
    for( int i = 0; i < 3600; i++ )
    {
        if( out[i] != 60 )
        {
            num++;
        }
    }

    printf( "%f\n", out[0] );
    printf( "%f\n", out[1] );
    printf( "%f\n", out[2] );
    printf( "%f\n", out[3599] );
    //printf( "%f\n", out[3600] );

    printf( "%f\n", 3333333333333.0 );
    printf( "%f\n", num );

    return 0;
}