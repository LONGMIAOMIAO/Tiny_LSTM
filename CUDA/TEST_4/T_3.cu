
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>

// template <typename T>
// __global__
// void cal( T* left, T* out )
// {
//     const int i = blockIdx.x * blockDim.x + threadIdx.x;
//     out[i] = left[i] * 3;
// }

template <typename T>
__global__
void cal( T* left, T* out, T* w, int left_Row, int left_Col, int w_Row, int w_Col )
{
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    T sum = 0;
    for ( int i = 0; i < left_Col; ++i )
    {
        sum += left[ row * left_Col + i ] * w[ i * w_Col + col ];
    } 
    out[ row * w_Col + col ] = sum;
}

int main()
{
    float* left;
    float* w;
    float* out;

    hipMallocManaged( &left, 1 * 6 * sizeof(float) );
    hipMallocManaged( &w, 1 * 6 * sizeof(float) );
    hipMallocManaged( &out, 1 * 4 * sizeof(float) );

    left[0] = 0.1;
    left[1] = 0.2;
    left[2] = 0.3;
    left[3] = 0.4;
    left[4] = 0.5;
    left[5] = 0.6;

    w[0] = 1;
    w[1] = 2;
    w[2] = 3;
    w[3] = 4;
    w[4] = 5;
    w[5] = 6;

    dim3 DimGrid(1, 1, 1);
    dim3 DimBlock(2, 2, 1);

    cal<<< DimGrid, DimBlock >>>( left, out, w, 2, 3, 3, 2 );
    hipDeviceSynchronize();

    printf( "%f\n", out[0] );
    printf( "%f\n", out[1] );
    printf( "%f\n", out[2] );
    printf( "%f\n", out[3] );

    return 0;
}


// // Compute C = A * B , Matrix C = hA * wB = rowA * columnB
// __global__ void matrixMultiply(float *A, float *B, float *C, int numARows,
//                                int numAColumns, int numBRows, int numBColumns,
//                                int numCRows, int numCColumns) {
//   //@@ Insert code to implement matrix multiplication here
//      float sum = 0.0f;

//     int row = blockIdx.y*blockDim.y + threadIdx.y;
//     int col = blockIdx.x*blockDim.x + threadIdx.x;


//     if(row < numCRows && col < numCColumns){
//         for (int i = 0; i < numAColumns; ++i)
//         {
//             sum += A[row*numAColumns + i] * B[i*numBColumns + col];
//         }
//         C[row*numBColumns + col] = sum;
//     }
//     printf("C = %f\n",C[row*numBColumns + col]);

// }