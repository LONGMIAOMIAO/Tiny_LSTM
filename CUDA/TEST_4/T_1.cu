
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>
__global__ 
void cal_1( double* in, double* out )
{
    const int i = blockIdx.x * blockDim.x + threadIdx.x;
    out[i] = sqrt(sqrt(sqrt( in[i] )));
    //out[i] = in[i];
}

int main()
{
    double* in;
    double* out;

    hipMallocManaged( &in, 1024 * 256 * sizeof(double) );
    hipMallocManaged( &out, 1024 * 256 * sizeof(double) );

    for( int i = 0; i < 1024 * 256; i++ )
    {
        in[i] = i;
    }
    
    cal_1<<< 1024, 256 >>>( in, out );

    hipDeviceSynchronize();

    double sum = 0;
    for( int i = 0; i < 1024 * 256; i++ )
    {
        sum += out[i];
    }
    printf( "%f\n", sum );
    
    double sum_2 = 0;
    for( int i = 0; i < 1024 * 256; i++ )
    {
        sum_2 += sqrt(sqrt(sqrt(in[i])));
    }
    printf( "%f\n", sum_2 );

    hipFree(in);
    hipFree(out);
    return 0;
}