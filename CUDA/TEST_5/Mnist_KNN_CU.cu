
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>
#include <fstream>  
#include <sstream>  
#include <iostream>
#include <map>
#include <time.h>

template <typename T>
struct Mat
{
    int width;
    int height;
    T* elements;
};

template <typename T>
__device__ 
T getElement( Mat<T> *A, int row, int col )
{
    return A->elements[ row * A->width + col ];
}

template <typename T>
__device__
void setElement( Mat<T>* A, int row, int col, T value )
{
    A->elements[ row * A->width + col ] = value;
}

template <typename T>
__global__
void matMulKernel(Mat<T> *A, Mat<T> *B, Mat<T> *C)
{
    T cvalue = 0;

	int row = threadIdx.y + blockIdx.y * blockDim.y;
	int col = threadIdx.x + blockIdx.x * blockDim.x;
	for (int i = 0; i < A->width; ++i)
	{
		cvalue += getElement(A, row, i) * getElement(B, i, col);
	}
	setElement(C, row, col, cvalue);
}

//  =============================================================================================

template <typename T>
__device__ 
T cal( Mat<T>* A, Mat<T>* B, int col, int i)
{
    //return A->elements[ row * A->width + col ];
    T val = 0;

    for( int j = 0; j < 784; ++j )
    {
        val += abs( getElement(A, col, j) - getElement(B, i, j) ); 
    }
    return val;
}

template <typename T>
__global__
void calDistance( Mat<T>* A, Mat<T>* B, int i, Mat<T>* C )
{
    int col = threadIdx.x + blockIdx.x * blockDim.x;

    C->elements[col] = cal<T>( A, B, col, i );
    //C->elements[col] = 3.0;
}



template <typename T>
void loadMnist( Mat<T>*& mat_L_Tr, Mat<T>*& mat_R_Tr, Mat<T>* mat_L_Te, Mat<T>* mat_R_Te )
{
    std::ifstream L_Tr_File( "../../../DATA/MNIST/train.csv", std::ios::in );
    std::string L_Tr_Str;
    int L_Tr_Num = 0;
    while (std::getline(L_Tr_File, L_Tr_Str))
    {
        //  printf( "121" );
        std::stringstream ss(L_Tr_Str);
        std::string str;
        int inNum = 0;
        while (std::getline(ss, str, ','))
        {
            //  ORIGDATA::mat_L_Tr_2D->data_f(L_Tr_Num,inNum) = atoi( str.c_str() );
            mat_L_Tr->elements[ L_Tr_Num * 784 + inNum ] = atoi( str.c_str() );
            //  printf( "%f", atof( str.c_str() ) );
            //  printf( "\n" );
            inNum++;
        }
        L_Tr_Num++;
    }

    std::ifstream R_Tr_File( "../../../DATA/MNIST/trainL.csv", std::ios::in );
    std::string R_Tr_Str;
    int R_Tr_Num = 0;
    while (std::getline(R_Tr_File, R_Tr_Str))
    {
        std::stringstream ss(R_Tr_Str);
        std::string str;
        int inNum = 0;
        while (std::getline(ss, str, ','))
        {
            //  ORIGDATA::mat_R_Tr_2D->data_b(R_Tr_Num,inNum) = atoi( str.c_str() );
            mat_R_Tr->elements[ R_Tr_Num * 10 + inNum ] = atoi( str.c_str() );
            //  printf( "%f", atof( str.c_str() ) );
            //  printf( "\n" );
            inNum++;
        }
        R_Tr_Num++;
    }

    std::ifstream L_Te_File( "../../../DATA/MNIST/test.csv", std::ios::in );
    std::string L_Te_Str;
    int L_Te_Num = 0;
    while (std::getline(L_Te_File, L_Te_Str))
    {
        std::stringstream ss(L_Te_Str);
        std::string str;
        int inNum = 0;
        while (std::getline(ss, str, ','))
        {
            //  ORIGDATA::mat_L_Te_2D->data_f(L_Te_Num,inNum) = atoi( str.c_str() );
            mat_L_Te->elements[ L_Te_Num * 784 + inNum ] = atoi( str.c_str() );
            inNum++;
        }
        L_Te_Num++;
    }

    std::ifstream R_Te_File( "../../../DATA/MNIST/testL.csv", std::ios::in );
    std::string R_Te_Str;
    int R_Te_Num = 0;
    while (std::getline(R_Te_File, R_Te_Str))
    {
        std::stringstream ss(R_Te_Str);
        std::string str;
        int inNum = 0;
        while (std::getline(ss, str, ','))
        {
            //  ORIGDATA::mat_R_Te_2D->data_b(R_Te_Num,inNum) = atoi( str.c_str() );
            mat_R_Te->elements[ R_Te_Num * 10 + inNum ] = atoi( str.c_str() );            
            inNum++;
        }
        R_Te_Num++;
    }
}


//  nvcc Mnist_KNN_CU.cu -O3
//  72s     96% correct
int main()
{
    auto start = clock();

    Mat<float>* mat_L_Tr;
    Mat<float>* mat_R_Tr;
    Mat<float>* mat_L_Te;
    Mat<float>* mat_R_Te;
    Mat<float>* mat_C;


    hipMallocManaged( &mat_L_Tr, sizeof(Mat<float>) );
    hipMallocManaged( &mat_R_Tr, sizeof(Mat<float>) );
    hipMallocManaged( &mat_L_Te, sizeof(Mat<float>) );
    hipMallocManaged( &mat_R_Te, sizeof(Mat<float>) );  
    hipMallocManaged( &mat_C, sizeof(Mat<float>) );  

    hipMallocManaged( &mat_L_Tr->elements, 55000 * 784 * sizeof(float) );
    hipMallocManaged( &mat_R_Tr->elements, 55000 * 10  * sizeof(float) );
    hipMallocManaged( &mat_L_Te->elements, 10000 * 784 * sizeof(float) );
    hipMallocManaged( &mat_R_Te->elements, 10000 * 10  * sizeof(float) );
    hipMallocManaged( &mat_C->elements, 55000 * sizeof(float) );

    mat_L_Tr->width     =    784;
    mat_L_Tr->height    =    55000;

    mat_R_Tr->width     =    10;
    mat_R_Tr->height    =    55000;

    mat_L_Te->width     =    784;
    mat_L_Te->height    =    10000;

    mat_R_Te->width     =    10;
    mat_R_Te->height    =    10000;

    mat_C->width    =   55000;
    mat_C->height   =   1;


    loadMnist<float>( mat_L_Tr, mat_R_Tr, mat_L_Te, mat_R_Te );

    dim3 DimGrid(55, 1, 1);
    dim3 DimBlock(1000, 1, 1);

    int totalNum = 0;
    for( int i = 0; i < 10000; i++ )
    {
        calDistance<float><<< DimGrid, DimBlock >>>( mat_L_Tr, mat_L_Te, i, mat_C );
        hipDeviceSynchronize();

        std::pair<float,int> min_Pair;
        min_Pair.first = mat_C->elements[0];
        for( int j = 0; j < 55000; j++ )
        {
            if( mat_C->elements[j] <= min_Pair.first )
            {
                min_Pair.first = mat_C->elements[j];
                min_Pair.second = j;
            }    
        }
        // std::map<float, int> m_map;
        // for( int j = 0; j < 55000; j++ )
        // {
        //     m_map[mat_C->elements[j]] = j;
        // }
        // auto top_K = m_map.begin();
        // int seq = (*top_K).second;
        int seq = min_Pair.second;

        int r_val = -1;
        for( int k = 0; k < 10; k++ )
        {
            //if ( getElement(mat_R_Tr, seq, k) == 1 )
            if ( mat_R_Tr->elements[ seq * 10 + k ] == 1 )
            {
                r_val = k;
                break;
            }        
        }

        int l_val = -2;
        for( int k = 0; k < 10; k++ )
        {
            //if ( getElement(mat_R_Te, i, k) == 1 )
            if ( mat_R_Te->elements[ i * 10 + k ] == 1 )
            {
                l_val = k;
                break;
            }        
        }
        if( r_val == l_val )
        {
            totalNum++;
        }

        // std::map<float, int> m_map;
        // float s = mat_C->elements[0];
        // for( int j = 0; j < 55000; j++ )
        // {
        //     if( mat_C->elements[j] < s )
        //     {
        //         s = mat_C->elements[j];
        //     }
        //     m_map[mat_C->elements[j]] = j;
        // }

        // int v = 0;
        // for( auto u = m_map.begin(); u != m_map.end(); u++ )
        // {
        //     if(v==10) break;

        //     printf("%d", (*u).second );
        //     printf( "\n" );
        //     v++;
        // }
        
        // printf( "%f", s );
        // printf( "\n" );
    }



    // void calDistance( Mat<T>*& A, Mat<T>*& B, int i, Mat<T>*& C )
    // {
    // int col = threadIdx.x + blockIdx.x * blockDim.x;

    // C->elements[col] = cal<T>( A, B, col, i );
    // }

    //matMulKernel<float><<< DimGrid, DimBlock >>>( left, w, out );
    printf( "%d", totalNum );
    printf( "\n" );
    auto end = clock();
    printf( "%lf", (end - start) / 1000000.0 );
    return 0;
}