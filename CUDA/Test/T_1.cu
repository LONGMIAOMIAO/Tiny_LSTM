
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void sayHellow(void)
{
    printf("Hello Outer!\n");
}

//  nvcc -arch sm_30 T_1.cu
//  nvcc -arch sm_60 T_1.cu
int main()
{
    printf("Hellow World!\n");

    sayHellow<<<1, 10>>>();

    //cudaDeviceSynchronize();
    hipDeviceReset();
    
    return 0;
}